#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../impl_template/tmpl_etc.cu"

static __global__ void k__pourcent_btcusdt(
	float * somme, float * potentiel,
	float * y, float * p1p0,
	float coef_puissance,
	uint * ts__d,
	uint P, uint Y)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (t < GRAND_T && mega_t < MEGA_T) {
		uint ty        = t_MODE(t, mega_t);
		uint t_btcusdt = ts__d[t] + mega_t;
		//
		FOR(0, p, P) {
			uint a_t_il_predit = (sng(p1p0[t_btcusdt*P + p]) == sng(y[ty*Y + 1+p]));
			//
			float _____somme = powf(fabs(p1p0[t_btcusdt*P + p]), coef_puissance) * a_t_il_predit;
			float _potentiel = powf(fabs(p1p0[t_btcusdt*P + p]), coef_puissance) * true         ;
			//
			atomicAdd(&somme    [p], _____somme);
			atomicAdd(&potentiel[p], _potentiel);
		}
	}
};

float *  pourcent_btcusdt(BTCUSDT_t * btcusdt, float * y__d, uint * ts__d, float coef_puissance) {
	uint P = btcusdt->Y;
	uint Y = btcusdt->P + btcusdt->A;
	//
	float *     somme__d = cudalloc<float>(P);
	float * potentiel__d = cudalloc<float>(P);
	//
	k__pourcent_btcusdt<<<dim3(KERD(GRAND_T, 16), KERD(MEGA_T, 8)), dim3(16,8)>>>(
		somme__d, potentiel__d,
		y__d, btcusdt->sorties__d,
		coef_puissance,
		ts__d,
		P, Y
	);
	ATTENDRE_CUDA();
	//
	float * somme     = gpu_vers_cpu<float>(    somme__d, P);
	float * potentiel = gpu_vers_cpu<float>(potentiel__d, P);
	//
	float * ret = alloc<float>(P);
	FOR(0, p, P) ret[p] = somme[p] / potentiel[p];
	//
	cudafree<float>(    somme__d);
	cudafree<float>(potentiel__d);
	    free(           somme   );
	    free(       potentiel   );
	//
	return ret;
};
