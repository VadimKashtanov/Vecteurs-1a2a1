#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../impl_template/tmpl_etc.cu"

static __global__ void k__f_btcusdt(
	float * somme_score,
	float * y, float * p1p0,
	uint * ts__d,
	uint P, uint Y,
	uint T)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (t < GRAND_T && mega_t < MEGA_T) {
		uint ty        = t_MODE(t, mega_t);
		uint t_btcusdt = ts__d[t] + mega_t;
		assert(t_btcusdt < T);
		//
		float A = y[ty*Y + 0];
		assert(A >= 0 && A <= +1);
		//
		FOR(0, p, P) {
			float _y = y[ty*Y + 1+p];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*P + p];
			//
			float _S = S(A, _y, _p1p0);
			

			assert(_S >= 0);


			//printf(">> %f %f\n", _y, p1p0[t_btcusdt*P + p]);
			//if (t==0 && mega_t==0 && p==0) printf("%f %f\n", _y, _p1p0);
			//
			atomicAdd(&somme_score[0], _S);
		}
	}
};

float f_btcusdt(BTCUSDT_t * btcusdt, float * y__d, uint * ts__d) {
	uint P = btcusdt->P;
	uint Y = btcusdt->P + btcusdt->A;
	//
	float * somme__d = cudalloc<float>(1);
	//
	k__f_btcusdt<<<dim3(KERD(GRAND_T, 16), KERD(MEGA_T, 8)/*, KERD(P, 4)*/), dim3(16,8/*,4*/)>>>(
		somme__d,
		y__d, btcusdt->sorties__d,
		ts__d,
		P, Y,
		btcusdt->T
	);
	ATTENDRE_CUDA();
	//
	float * somme = gpu_vers_cpu<float>(somme__d, 1);
	//
	float score = somme[0] / ((float)(P * GRAND_T * MEGA_T));
	//
	cudafree<float>(somme__d);
	    free       (somme   );
	//
	return score;
};