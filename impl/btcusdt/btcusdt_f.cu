#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../impl_template/tmpl_etc.cu"

static __global__ void k__f_btcusdt__moyenneK(
	float * moyenneK,
	float * y, float * p1p0,
	uint * ts__d,
	uint P, uint Y,
	uint T)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (t < GRAND_T && mega_t < MEGA_T) {
		uint ty        = t_MODE(t, mega_t);
		uint t_btcusdt = ts__d[t] + mega_t;
		assert(t_btcusdt < T);
		//
		FOR(0, p, P) {
			float _y = y[ty*Y + 1+p];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*P + p];
			//
			float _k = K(_y, _p1p0) / (GRAND_T*MEGA_T);
			//
			atomicAdd(&moyenneK[p], _k);
		}
	}
};

static __global__ void k__f_btcusdt(
	float * somme_score,
	float * moyenneK,
	float * y, float * p1p0,
	uint * ts__d,
	uint P, uint Y,
	uint T)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (t < GRAND_T && mega_t < MEGA_T) {
		uint ty        = t_MODE(t, mega_t);
		uint t_btcusdt = ts__d[t] + mega_t;
		assert(t_btcusdt < T);
		//
		float A = y[ty*Y + 0];
		assert(A >= -1 && A <= +1);
		//
		FOR(0, p, P) {
			float _y = y[ty*Y + 1+p];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*P + p];
			//
			float _S = S(A, _y, _p1p0);
			assert(_S >= 0);
			//
			atomicAdd(&somme_score[0], _S / moyenneK[p]);
		}
	}
};

float f_btcusdt(BTCUSDT_t * btcusdt, float * y__d, uint * ts__d) {
	uint P = btcusdt->P;
	uint Y = btcusdt->P + btcusdt->A;
	//
	//
	float * moyenneK__d = cudalloc<float>(P);
	k__f_btcusdt__moyenneK<<<dim3(KERD(GRAND_T, 16), KERD(MEGA_T, 8)), dim3(16,8)>>>(
		moyenneK__d,
		y__d, btcusdt->sorties__d,
		ts__d,
		P, Y,
		btcusdt->T
	);
	ATTENDRE_CUDA();
	//
	//
	float * somme__d = cudalloc<float>(1);
	k__f_btcusdt<<<dim3(KERD(GRAND_T, 16), KERD(MEGA_T, 8)), dim3(16,8)>>>(
		somme__d,
		moyenneK__d,
		y__d, btcusdt->sorties__d,
		ts__d,
		P, Y,
		btcusdt->T
	);
	ATTENDRE_CUDA();
	//
	//
	float * somme = gpu_vers_cpu<float>(somme__d, 1);
	//
	float score = somme[0] / ((float)(P * GRAND_T * MEGA_T));
	//
	//
	cudafree<float>(somme__d   );
	cudafree<float>(moyenneK__d);
	    free       (somme      );
	//
	return score;
};