#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../impl_template/tmpl_etc.cu"

static __global__ void k__df_btcusdt(
	float * y, float * p1p0, float * dy,
	uint * ts__d,
	uint P, uint Y)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (t < GRAND_T && mega_t < MEGA_T) {
		uint ty        = t_MODE(t, mega_t);
		uint t_btcusdt = ts__d[t] + mega_t;
		//
		float A = y[ty*Y + 0];
		assert(A >= 0 && A <= +1);
		float _da = 0;
		//
		FOR(0, p, P) {
			float _y = y[ty*Y + 1+p];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*P + p];
			//
			//
			float _dSdy = dSdy(A, _y, _p1p0) / (float)(P * MEGA_T * GRAND_T);
			float _dSdA = dSdA(A, _y, _p1p0) / (float)(P * MEGA_T * GRAND_T);
			//
			//printf("%f %f\n", _y, p1p0[t_btcusdt*P + p]);
			//if (t==0 && mega_t==0 && p==0) printf("%f %f\n", _y, _p1p0);
			//
			atomicAdd(&dy[ty*Y + 1+p], _dSdy);
			_da += _dSdA;
		}
		//
		atomicAdd(&dy[ty*Y + 0], _da);
	}
};

void df_btcusdt(BTCUSDT_t * btcusdt, float * y__d, float * dy__d, uint * ts__d) {
	uint P = btcusdt->P;
	uint Y = btcusdt->P + btcusdt->A;
	//
	k__df_btcusdt<<<dim3(KERD(GRAND_T, 16), KERD(MEGA_T, 8)), dim3(16,8)>>>(
		y__d, btcusdt->sorties__d, dy__d,
		ts__d,
		P, Y
	);
	ATTENDRE_CUDA();
};