#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../impl_template/tmpl_etc.cu"

static __global__ void k__f_btcusdt__moyenneK(
	float * moyenneK,
	float * y, float * p1p0,
	uint * ts__d,
	uint P, uint Y,
	uint T)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (t < GRAND_T && mega_t < MEGA_T) {
		uint ty        = t_MODE(t, mega_t);
		uint t_btcusdt = ts__d[t] + mega_t;
		assert(t_btcusdt < T);
		//
		FOR(0, p, P) {
			float _y = y[ty*Y + 1+p];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*P + p];
			//
			float _k = K(_y, _p1p0) / (GRAND_T*MEGA_T);
			//
			atomicAdd(&moyenneK[p], _k);
		}
	}
};

static __global__ void k__df_btcusdt(
	float * moyenneK,
	float * y, float * p1p0, float * dy,
	uint * ts__d,
	uint P, uint Y)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (t < GRAND_T && mega_t < MEGA_T) {
		uint ty        = t_MODE(t, mega_t);
		uint t_btcusdt = ts__d[t] + mega_t;
		//
		float A = y[ty*Y + 0];
		assert(A >= -1 && A <= +1);
		float _da = 0;
		//
		FOR(0, p, P) {
			float _y = y[ty*Y + 1+p];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*P + p];
			//
			//
			float _dSdy = dSdy(A, _y, _p1p0) / (float)(P * MEGA_T * GRAND_T) / moyenneK[p];
			float _dSdA = dSdA(A, _y, _p1p0) / (float)(P * MEGA_T * GRAND_T) / moyenneK[p];
			//
			atomicAdd(&dy[ty*Y + 1+p], _dSdy);
			_da += _dSdA;
		}
		//
		atomicAdd(&dy[ty*Y + 0], _da);
	}
};

void df_btcusdt(BTCUSDT_t * btcusdt, float * y__d, float * dy__d, uint * ts__d) {
	uint P = btcusdt->P;
	uint Y = btcusdt->P + btcusdt->A;
	//
	//
	float * moyenneK__d = cudalloc<float>(P);
	k__f_btcusdt__moyenneK<<<dim3(KERD(GRAND_T, 16), KERD(MEGA_T, 8)), dim3(16,8)>>>(
		moyenneK__d,
		y__d, btcusdt->sorties__d,
		ts__d,
		P, Y,
		btcusdt->T
	);
	ATTENDRE_CUDA();
	//
	//
	k__df_btcusdt<<<dim3(KERD(GRAND_T, 16), KERD(MEGA_T, 8)), dim3(16,8)>>>(
		moyenneK__d,
		y__d, btcusdt->sorties__d, dy__d,
		ts__d,
		P, Y
	);
	ATTENDRE_CUDA();
	//
	//
	cudafree<float>(moyenneK__d);
};