#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_template/tmpl_etc.cu"

static void cree_mdl_depuis_pre_mdl(BTCUSDT_t * btcusdt) {
	Mdl_t * mdl = cree_mdl_depuis_st_bin("mdl.st.bin");
	mdl_verif(mdl, btcusdt);
	ecrire_mdl("mdl.bin", mdl);
	liberer_mdl(mdl);
};

static void plumer_le_score(Mdl_t * mdl, BTCUSDT_t * btcusdt) {
	uint T = btcusdt->T-1;
	T = T - (T%(GRAND_T * MEGA_T));
	T = T / (GRAND_T * MEGA_T);
	//
	float p0[btcusdt->Y]; FOR(0, i, btcusdt->Y) p0[i] = 0.0;
	float p1[btcusdt->Y]; FOR(0, i, btcusdt->Y) p1[i] = 0.0;
	float p3[btcusdt->Y]; FOR(0, i, btcusdt->Y) p3[i] = 0.0;
	float p8[btcusdt->Y]; FOR(0, i, btcusdt->Y) p8[i] = 0.0;
	//
	FOR(0, _t_, T) {
		uint ts[GRAND_T];
		FOR(0, t, GRAND_T) ts[t] = _t_*GRAND_T*MEGA_T + t*MEGA_T;
		//
		uint * ts__d = cpu_vers_gpu<uint>(ts, GRAND_T);
		//
		float * _p0 = mdl_pourcent(mdl, btcusdt, ts__d, 0.0);
		float * _p1 = mdl_pourcent(mdl, btcusdt, ts__d, 1.0);
		float * _p3 = mdl_pourcent(mdl, btcusdt, ts__d, 3.0);
		float * _p8 = mdl_pourcent(mdl, btcusdt, ts__d, 8.0);
		//
		cudafree<uint>(ts__d);
		//
		FOR(0, i, btcusdt->Y) p0[i] += _p0[i] / (float)T;
		FOR(0, i, btcusdt->Y) p1[i] += _p1[i] / (float)T;
		FOR(0, i, btcusdt->Y) p3[i] += _p3[i] / (float)T;
		FOR(0, i, btcusdt->Y) p8[i] += _p8[i] / (float)T;
		//
		free(_p0);
		free(_p1);
		free(_p3);
		free(_p8);
	};
	//
	FOR(0, i, btcusdt->Y) {
		printf("\033[93mPRED MODEL[%i]\033[0m : \033[96m%f%%\033[0m (^1=\033[96m%f%%\033[0m ^3=\033[96m%f%%\033[0m ^8=\033[96m%f%%\033[0m)\n",
			i,
			p0[i],
			p1[i],
			p3[i],
			p8[i]
		);
	}
};

void visualiser_vitesses(char * mdl_bin, BTCUSDT_t * btcusdt) {
	//	Sans que ça soit optimisé
	Mdl_t * mdl = ouvrire_mdl(mdl_bin);
	//
	mdl_desoptimiser(mdl);
	//
	uint ts[GRAND_T];
	FOR(0, t, GRAND_T)
		ts[t] = rand() % (btcusdt->T - MEGA_T - 1);
	uint * ts__d = cpu_vers_gpu<uint>(ts, GRAND_T);
	//
	float * temps = mdl_allez_retour_temps(mdl, btcusdt, ts__d);
	//
	temps; // INSTS + INSTS (F et F')
	//
	//
	printf(" --- Temps F(x) ---\n");
	float _max = max_lst<float>(temps+0*mdl->BLOQUES, mdl->BLOQUES);
	printf("temp max = %f\n", _max);
	FOR(0, i, mdl->BLOQUES) {
		uint pts = (uint)roundf(30.0*temps[0+mdl->BLOQUES + i] / _max);
		printf("%4.i| ", i);
		FOR(0, j, pts) printf("\033[103m_\033[0m");
		FOR(pts, j, 30) printf(" ");
		printf("  %s\n", inst_Nom[mdl->inst[i]->ID]);
	}
	//
	//
	printf(" --- Temps dF(x) ---\n");
	_max = max_lst<float>(temps+1*mdl->BLOQUES, mdl->BLOQUES);
	printf("temp max = %f\n", _max);
	FOR(0, i, mdl->BLOQUES) {
		uint pts = (uint)roundf(30.0*temps[1+mdl->BLOQUES + i] / _max);
		printf("%4.i| ", i);
		FOR(0, j, pts) printf("\033[104m_\033[0m");
		FOR(pts, j, 30) printf(" ");
		printf("  %s\n", inst_Nom[mdl->inst[i]->ID]);
	}
	//
	free(temps);
	//
	//
	liberer_mdl(mdl);
};

void montrer_Y_du_model(Mdl_t * mdl, BTCUSDT_t * btcusdt) {
	uint ts[GRAND_T];
	FOR(0, t, GRAND_T)
		ts[t] = rand() % (btcusdt->T - MEGA_T - 1);
	uint * ts__d = cpu_vers_gpu<uint>(ts, GRAND_T);
	//
	mdl_allez_retour(mdl, btcusdt, ts__d);
	//
	printf(" ======= Plumer Y ======\n");
	printf("mega_t = | ");
	FOR(0, i, MIN2(MEGA_T, 19)) printf("    %i   |", i);
	printf("\n");
	FOR(0, i, mdl->insts)
	{
		Inst_t * inst = mdl->inst[i];
		printf("#%i -- ID=%i %s Y=%i --\n", i, inst->ID, inst_Nom[inst->ID], inst->Y);
		//
		float * y = gpu_vers_cpu<float>(inst->y__d, inst->Y * GRAND_T * MEGA_T);
		//
		FOR(0, j, inst->Y) {
			printf("%i| ", j);
			FOR(0, mega_t, MIN2(MEGA_T, 19)) {
				printf("%+f ", y[mega_t*GRAND_T*inst->Y + 0*inst->Y + j]);
			}
			printf("\n");
		}
		//
		free(y);
	};
	//
	cudafree<uint>(ts__d);
};

int main() {
	srand(time(NULL));
	ecrire_structure_generale("structure_generale.bin");
	verif_insts();

	//	=========================================================
	//	=========================================================
	//	=========================================================
	//verif_mdl_1e5();

	//exit(0);

	//	=========================================================
	//	=========================================================
	//	=========================================================
	BTCUSDT_t * btcusdt = cree_btcusdt("prixs/dar.bin");
	MSG("Kconvl f & df optimisée (Important)");
	MSG("Pool2d optimisé (f et df)");
	//
	MSG("ADAM n'est pas utilisé");
	//
	MSG("P du model peut etre changé. P=3 par exemple")

	//	=========================================================
	//	=========================================================
	//	=========================================================

	//visualiser_vitesses("mdl.bin", btcusdt);

	//	=========================================================
	//	=========================================================
	//	=========================================================

	//	--- Re-cree le Model ---
	cree_mdl_depuis_pre_mdl(btcusdt);

	//	--- Mdl_t ---
	Mdl_t * mdl = ouvrire_mdl("mdl.bin");
	plumer_model(mdl);
	//montrer_Y_du_model(mdl, btcusdt);
	//tester_le_model(mdl, btcusdt);

	//	=========================================================
	//	=========================================================
	//	=========================================================
	uint un_mois = ((24*30 - (24*30 % MEGA_T)) / MEGA_T) * MEGA_T;
	//
//plumer_le_score(mdl, btcusdt);
	// 
	uint e = 0;       // Atention Mechanisme, alternative Dot1d AB, ...
	while (true) {
		printf(" === Echope %i ===\n", e);
		
		//
		uint I        = 10;
		uint tous_les = 10;
		
		//
		srand(time(NULL));
		uint ts[GRAND_T];
		FOR(0, t, GRAND_T)
			ts[t] = rand() % (btcusdt->T - MEGA_T - 1 - un_mois);
		uint * ts__d = cpu_vers_gpu<uint>(ts, GRAND_T);

		//
		opti(
			mdl, btcusdt,
			ts__d,
			I,
			tous_les,
			SGD, 3e-5
		);
		ecrire_mdl("mdl.bin", mdl);

		if (e % 10 == 0) {
			printf("pause ...\n");
			sleep(2);
		}
		
		//
		if (e % 50 == 0 && e != 0) {
			plumer_le_score(mdl, btcusdt);
		}
		e++;

		//
		cudafree<uint>(ts__d);
	}

	//
	//liberer_mdl    (mdl    );
	//liberer_btcusdt(btcusdt);
};






/*
Bon parametres:

6 juin 2024 = grand=10*16, mega_t=24, alpha=5e-4, I=50, L2=0.0, K=(sng==sng ? 0.25 : 2.0)

7 juin 2024 :
	grand_t = 3*16
	mega_t  = 24
	alpha = 5e-4
	I = 40
	L2=0.0
	K=(0.25 : 2.0)
	model = N=8 max_interv=256
		kconvl_lstm+pool -> chaine x10                   -> chaine -> x10
					-> kconvl_lstm+pool -> chaine -> x10 ->

*/