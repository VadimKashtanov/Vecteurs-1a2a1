#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../impl_template/tmpl_etc.cu"

uint hists[] = {
	ADAM_HISTOIRE
};

void opti(
	Mdl_t     *     mdl,
	BTCUSDT_t * btcusdt,
	uint      *   ts__d,
	uint              I,
	uint       tous_les,
	uint        methode,
	float         alpha
) {
	//	--- Hist ---
	float *** hist = alloc<float**>(hists[methode]);
	FOR(0, h, hists[methode]) {
		hist[h] = alloc<float*>(mdl->insts);
		FOR(0, i, mdl->insts) {
			hist[h][i] = cudalloc<float>(mdl->inst[i]->P);
			// = 0
		}
	}
	//	--- Plume ---
	mdl_plume_grad(mdl, btcusdt, ts__d);
	//
	float _max_abs_grad = 1;//mdl_max_abs_grad(mdl);
	if (_max_abs_grad == 0) ERR("Le grad max est = 0");
	//
	printf("alpha=%f, max_abs_grad=%f => nouveau alpha=%f\n", alpha, _max_abs_grad, alpha / _max_abs_grad);
	//
	//	--- Opti  ---
	FOR(0, i, I) {
		/*uint alea_ts[GRAND_T];
		FOR(0, j, GRAND_T) alea_ts[j] = rand() % (btcusdt->T - MEGA_T - 1);
		CONTROLE_CUDA(hipMemcpy(ts__d, alea_ts, sizeof(uint)*GRAND_T, hipMemcpyHostToDevice));*/

		if (i != 0) {
			//	dF(x)
			mdl_allez_retour(mdl, btcusdt, ts__d);
			//	x = x - dx
			if (methode == ADAM) adam(mdl, hist, i, alpha / _max_abs_grad);
		}
		//
		if (i % tous_les == 0) {
			float s = mdl_S(mdl, btcusdt, ts__d);
			float * p0 = mdl_pourcent(mdl, btcusdt, ts__d, 0.0);
			float * p1 = mdl_pourcent(mdl, btcusdt, ts__d, 1.0);
			float * p8 = mdl_pourcent(mdl, btcusdt, ts__d, 4.0);
			//

			printf("%3.i/%3.i score = %f (", i, I, s);

			printf("^0:{");
			FOR(0, p, btcusdt->Y) printf("\033[96m%f%%\033[0m ", p0[p]);
			printf("}  ");

			printf("^1:{");
			FOR(0, p, btcusdt->Y) printf("\033[96m%f%%\033[0m ", p1[p]);
			printf("}  ");

			printf("^4:{");
			FOR(0, p, btcusdt->Y) printf("\033[96m%f%%\033[0m ", p8[p]);
			printf("}");

			printf(")\n");

			free(p0);
			free(p1);
			free(p8);
		};
	};
	//
	FOR(0, h, hists[methode]) {
		FOR(0, i, mdl->insts) {
			cudafree<float>(hist[h][i]);
		}
		free(hist[h]);
	}
	free(hist);
}