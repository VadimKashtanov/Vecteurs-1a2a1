#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_template/tmpl_etc.cu"

__global__
static void kerd_lire(float * p__d, uint p, float * val) {
	val[0] = p__d[p];
};

static float lire(float * p__d, uint p) {
	float * val = cudalloc<float>(1);
	kerd_lire<<<1,1>>>(p__d, p, val);
	ATTENDRE_CUDA();
	//
	float * _ret = gpu_vers_cpu<float>(val, 1);
	float ret = _ret[0];
	free(_ret);cudafree<float>(val);
	//
	return ret;
};

static float ** toutes_les_predictions(Mdl_t * mdl, BTCUSDT_t * btcusdt) {
	uint mode = 0;
	//
	uint T = 0;// = btcusdt->T;
	uint PREDS = 0;
	//
	if (GRAND_T*MEGA_T >= btcusdt->T) {
		T = btcusdt->T;
		T = T - (T % MEGA_T);
		T = T / MEGA_T;
		//
		PREDS = T * MEGA_T;
		//
		mode = 1;
	} else {
		T = btcusdt->T;
		T = T - (T%(GRAND_T * MEGA_T));
		T = T / (GRAND_T * MEGA_T);
		//
		PREDS = T * (GRAND_T * MEGA_T);
		//
		mode = 0;
	}
	//
	//uint partie_non_couverte = btcusdt->T - T*MEGA_T*GRAND_T;
	//
	float * les_Amplitudes  = alloc<float>(PREDS);
	float * les_predictions = alloc<float>(PREDS);
	float * les_deltas      = alloc<float>(PREDS);
	//
	uint lp = 0;
	//
	FOR(0, _t_, T) {
		uint ts[GRAND_T];
		//
		if (mode == 0) {
			FOR(0, t, GRAND_T) ts[t] = (btcusdt->T-PREDS) + _t_*GRAND_T*MEGA_T + t*MEGA_T;
		} else {
			FOR(0, t, GRAND_T) ts[t] = (btcusdt->T-PREDS) + _t_*MEGA_T + t*0;
		}
		//
		uint * ts__d = cpu_vers_gpu<uint>(ts, GRAND_T);
		//
		mdl_f(mdl, btcusdt, ts__d);
		//
		cudafree<uint>(ts__d);
		//
		float * y = gpu_vers_cpu<float>(mdl->inst[mdl->la_sortie]->y__d, GRAND_T*MEGA_T*(btcusdt->Y+1));
		//
		if (mode == 0) {
			FOR(0, t, GRAND_T) {
				FOR(0, mega_t, MEGA_T) {
					les_Amplitudes [lp]  = y[t*MEGA_T*(btcusdt->Y+1) + mega_t*(btcusdt->Y+1) + 0];
					les_predictions[lp]  = y[t*MEGA_T*(btcusdt->Y+1) + mega_t*(btcusdt->Y+1) + 1];
					les_deltas     [lp] = lire(btcusdt->sorties__d, (ts[t] + mega_t)*btcusdt->Y+0);
					if ((ts[t] + mega_t) >= btcusdt->T-1) MSG("(ts[t] + mega_t) == btcusdt->T-1\n");
					lp++;
				}
			}
		} else {
			uint t = 0;
			FOR(0, mega_t, MEGA_T) {
				les_Amplitudes [lp]  = y[t*MEGA_T*(btcusdt->Y+1) + mega_t*(btcusdt->Y+1) + 0];
				les_predictions[lp]  = y[t*MEGA_T*(btcusdt->Y+1) + mega_t*(btcusdt->Y+1) + 1];
				les_deltas     [lp] = lire(btcusdt->sorties__d, (ts[t] + mega_t)*btcusdt->Y+0);
				if ((ts[t] + mega_t) >= btcusdt->T-1) MSG("(ts[t] + mega_t) == btcusdt->T-1\n");
				lp++;
			}
		}
		//
		free(y);
	};
	//
	float ** ret = alloc<float*>(3);
	ret[0] = les_Amplitudes ;
	ret[1] = les_predictions;
	ret[2] = les_deltas     ;
	return ret;
};

int main() {
	srand(0);
	verif_insts();

	//	=========================================================
	//	=========================================================
	//	=========================================================
	BTCUSDT_t * btcusdt = cree_btcusdt("prixs/tester_model_donnee.bin");

	//	=========================================================
	//	=========================================================
	//	=========================================================

	//	--- Mdl_t ---
	Mdl_t * mdl = ouvrire_mdl("mdl.bin");
	//plumer_model(mdl);
	//montrer_Y_du_model(mdl, btcusdt);
	//tester_le_model(mdl, btcusdt);

	float ** __lp = toutes_les_predictions(mdl, btcusdt);
	float * A  = __lp[0];
	float * lp = __lp[1];
	float * dl = __lp[2];

	FILE * fp = FOPEN("les_predictions.bin", "wb");
	//
	uint T = 0;// = btcusdt->T;
	uint PREDS = 0;
	//
	if (GRAND_T*MEGA_T >= btcusdt->T) {
		T = btcusdt->T;
		T = T - (T % MEGA_T);
		T = T / MEGA_T;
		//
		PREDS = T * MEGA_T;
	} else {
		T = btcusdt->T;
		T = T - (T%(GRAND_T * MEGA_T));
		T = T / (GRAND_T * MEGA_T);
		//
		PREDS = T * (GRAND_T * MEGA_T);
	}
	//
	FWRITE(A, sizeof(float), PREDS, fp);	//les prédictions
	free(A);
	//
	FWRITE(lp, sizeof(float), PREDS, fp);	//les prédictions
	free(lp);
	//
	FWRITE(dl, sizeof(float), PREDS, fp);	//les déltas
	free(dl);
	//
	fclose(fp);

	//	=========================================================
	//	=========================================================
	//	=========================================================
	//
	//plumer_le_score(mdl, btcusdt);

	//
	liberer_mdl    (mdl    );
	liberer_btcusdt(btcusdt);
};