#include "hip/hip_runtime.h"
#include "dot1d_X.cuh"

template <uint BLOQUE>
static __global__ void d_kerd_f_ax_b__shared_16___dX(
	uint c0,
	//
	float * x0, uint X0_vars, uint X0, uint x0_t, float * dx0,
	//
	float *  y, uint Y__vars, uint  Y, float * dy,
	float *  l, uint L__vars,
	//
	float * p, float * dp,
	//
	uint mega_t,
	//
	uint v_x0, uint v_y, uint C0,
	//
	uint activ)
{
	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _x = thx + blockIdx.x * blockDim.x;
	//uint _tc0 = thy + blockIdx.y * blockDim.y;
	uint _t = thy + blockIdx.y * blockDim.y;

	//uint _t = _tc0 % GRAND_T;
	//uint c0 = (_tc0 - _t)/GRAND_T;
	
	uint depart_a0 = c0*(v_x0*v_y + v_y) + 0;
	uint depart__b = c0*(v_x0*v_y + v_y) + v_x0*v_y;
	uint depart__y = c0*v_y;
	uint depart__l = c0*v_y;
	uint depart_x0 = c0*v_x0;

 	//
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );

	float s = 0;

	FOR(0, d, Y/BLOQUE) {
		float _l  =  l[ty*L__vars + depart__l + (d*BLOQUE+thx)];
		float _dy = dy[ty*Y__vars + depart__y + (d*BLOQUE+thx)];
		__partage__x[thy][thx] =  _l * _dy;
		__partage__p[thy][thx] = p[depart_a0 + (d*BLOQUE+thy)*X0 + _x];
		__syncthreads();

#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

	atomicAdd(&dx0[tx0*X0_vars + depart_x0 + _x], s);
};

template <uint BLOQUE>
static __global__ void d_kerd_f_ax_b__shared_16___dA(
	uint c0,
	//
	float * x0, uint X0_vars, uint X0, uint x0_t, float * dx0,
	//
	float *  y, uint Y__vars, uint  Y, float * dy,
	float *  l, uint L__vars,
	//
	float * p, float * dp,
	//
	uint mega_t,
	//
	uint v_x0, uint v_y, uint C0,
	//
	uint activ)
{
	//dp = x.T @ ((y-_y)*dtanh(x@p))

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _x = thx + blockIdx.x * blockDim.x;
	//uint _xc0 = thx + blockIdx.x * blockDim.x;
	uint _y = thy + blockIdx.y * blockDim.y;

	//uint _x = _xc0 % X0;
	//uint c0 = (_xc0 - _x)/X0;
	
	uint depart_a0 = c0*(v_x0*v_y + v_y) + 0;
	uint depart__b = c0*(v_x0*v_y + v_y) + v_x0*v_y;
	uint depart__y = c0*v_y;
	uint depart__l = c0*v_y;
	uint depart_x0 = c0*v_x0;

 	//
	float s = 0;

	uint d = blockIdx.z;
	//FOR(0, d, T/BLOQUE) {
	//assert((d*BLOQUE+thy) < GRAND_T);
		uint tx0 = t_MODE((d*BLOQUE+thy), mega_t-x0_t);
		uint ty  = t_MODE((d*BLOQUE+thx), mega_t     );
		//
		float __l =  l[ty*L__vars + depart__l + _y];
		float _dy = dy[ty*Y__vars + depart__y + _y];
		__partage__x[thy][thx] = __l * _dy;
		__partage__p[thy][thx] = x0[tx0*X0_vars + depart_x0 + _x];
		__syncthreads();

	#pragma unroll
		FOR(0, i, BLOQUE) {
			s += __partage__x[thy][i] * __partage__p[i][thx];
		}
		__syncthreads();
	//};

	atomicAdd(&dp[depart_a0 + _y*X0 + _x], s);
};

template <uint BLOQUE>
static __global__ void d_kerd_f_ax_b__shared_16___db(
	uint c0,
	//
	float * x0, uint X0_vars, uint X0, uint x0_t, float * dx0,
	//
	float *  y, uint Y__vars, uint  Y, float * dy,
	float *  l, uint L__vars,
	//
	float * p, float * dp,
	//
	uint mega_t,
	//
	uint v_x0, uint v_y, uint C0,
	//
	uint activ)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	//uint _tc0 = threadIdx.y + blockIdx.y * blockDim.y;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;

	//uint _t = _tc0 % GRAND_T;
	//uint c0 = (_tc0 - _t)/GRAND_T;
	
	uint depart_a0 = c0*(v_x0*v_y + v_y) + 0;
	uint depart__b = c0*(v_x0*v_y + v_y) + v_x0*v_y;
	uint depart__y = c0*v_y;
	uint depart__l = c0*v_y;
	uint depart_x0 = c0*v_x0;

 	//
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );

	float _l  =  l[ty*L__vars + depart__l + _y];
	float _dy = dy[ty*Y__vars + depart__y + _y];
	atomicAdd(&dp[depart__b + _y], _l * _dy);
};

//	==========================================================

template <uint BLOQUE>
static __global__ void d_kerd_f_ax_b__t(
	uint c0,
	//
	float * x0, uint X0_vars, uint X0, uint x0_t, float * dx0,
	//
	float *  y, uint Y__vars, uint  Y, float * dy,
	float *  l, uint L__vars,
	//
	float * p, float * dp,
	//
	uint mega_t,
	//
	uint v_x0, uint v_y, uint C0,
	//
	uint activ)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y   = thx + blockIdx.x * blockDim.x;
	uint /*_tc0*/_t = thy + blockIdx.y * blockDim.y;

	//uint _t = _tc0 % GRAND_T;
	//uint c0 = (_tc0 - _t)/GRAND_T;

	uint depart_a0 = c0*(v_x0*v_y + v_y) + 0;
	uint depart__b = c0*(v_x0*v_y + v_y) + v_x0*v_y;
	uint depart__y = c0*v_y;
	uint depart__l = c0*v_y;
	uint depart_x0 = c0*v_x0;

	if (_y < Y && _t < GRAND_T) {
		//
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float _dy = dy[ty*Y__vars + depart__y + _y];
		float  _l =  l[ty*L__vars + depart__l + _y];
		//
		float ds = _dy * _l;
		//
		atomicAdd(&dp[depart__b + _y], ds);
		//
		FOR(0, i, v_x0) {
			//s += x0[tx0*X0_vars + depart_x0 + i] * p[depart_a0 + _y*X0 + i];
			atomicAdd(&dx0[tx0*X0_vars + depart_x0 + i], ds*p[depart_a0 + _y*X0 + i]);
			atomicAdd(&dp [depart_a0  + _y*X0      + i], ds*x0[tx0*X0_vars + depart_x0 + i]);
		}
	}
};

void dot1d_X__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	//
	uint \
		C0   =params[0], \
		activ=params[1];
	//
	uint v_x0 = inst->x_Y[0] / C0;
	uint v_y  = inst->Y      / C0;
	//
	uint x0_t = inst->x_t[0];
	uint Y    = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	uint xs_existants = x0_existe;
	//
	if (x0_existe) {
		if (v_y % 16 == 0 && GRAND_T % 16 == 0) {
			FOR(0, c0, C0) {
				d_kerd_f_ax_b__shared_16___dX<16><<<dim3(KERD(v_x0, 16), KERD(GRAND_T, 16)), dim3(16, 16)>>>(
					c0,
					//
					x__d[0], inst->x_Y[0], v_x0, x0_t, dx__d[0],
					//
					inst->y__d, inst->Y, v_y, inst->dy__d,
					inst->l__d, inst->L,
					//
					inst->p__d, inst->dp__d,
					//
					mega_t,
					//
					v_x0, v_y, C0,
					//
					activ);
				d_kerd_f_ax_b__shared_16___dA<16><<<dim3(KERD(v_x0, 16), KERD(v_y, 16), DIV(GRAND_T,16)), dim3(16, 16, 1)>>>(
					c0,
					//
					x__d[0], inst->x_Y[0], v_x0, x0_t, dx__d[0],
					//
					inst->y__d, inst->Y, v_y, inst->dy__d,
					inst->l__d, inst->L,
					//
					inst->p__d, inst->dp__d,
					//
					mega_t,
					//
					v_x0, v_y, C0,
					//
					activ);
				d_kerd_f_ax_b__shared_16___db<16><<<dim3(KERD(v_y, 16), KERD(GRAND_T,16)), dim3(16, 16)>>>(
					c0,
					//
					x__d[0], inst->x_Y[0], v_x0, x0_t, dx__d[0],
					//
					inst->y__d, inst->Y, v_y, inst->dy__d,
					inst->l__d, inst->L,
					//
					inst->p__d, inst->dp__d,
					//
					mega_t,
					//
					v_x0, v_y, C0,
					//
					activ);
			}
		} else {
			FOR(0, c0, C0) {
				d_kerd_f_ax_b__t<16><<<dim3(KERD(v_y, 16), KERD(GRAND_T, 16)), dim3(16, 16)>>>(
					c0,
					x__d[0], inst->x_Y[0], v_x0, x0_t, dx__d[0],
					//
					inst->y__d, inst->Y, v_y, inst->dy__d,
					inst->l__d, inst->L,
					//
					inst->p__d, inst->dp__d,
					//
					mega_t,
					//
					v_x0, v_y, C0,
					//
					activ);
			}
		}
	} else {
		//	rien
	}
};