#include "hip/hip_runtime.h"
#include "dot1d_X.cuh"

template <uint BLOQUE>
static __global__ void kerd_f_ax_b__t(
	uint c0,
	//
	float * x0, uint X0_vars, uint X0, uint x0_t,
	//
	float *  y, uint Y__vars, uint  Y,
	float *  l, uint L__vars,
	//
	float * p,
	//
	uint mega_t,
	//
	uint v_x0, uint v_y, uint C0,
	//
	uint activ)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y   = thx + blockIdx.x * blockDim.x;
	uint /*_tc0*/_t = thy + blockIdx.y * blockDim.y;

	//uint _t = _tc0 % GRAND_T;
	//uint c0 = (_tc0 - _t)/GRAND_T;

	uint depart_a0 = c0*(v_x0*v_y + v_y) + 0;
	uint depart__b = c0*(v_x0*v_y + v_y) + v_x0*v_y;
	uint depart__y = c0*v_y;
	uint depart__l = c0*v_y;
	uint depart_x0 = c0*v_x0;

	if (_y < Y) {
		//
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );

		float s = p[depart__b + _y];
		//
		FOR(0, i, v_x0) {
			s += x0[tx0*X0_vars + depart_x0 + i] * p[depart_a0 + _y*X0 + i];
		}
		//
		f2 a_da = ACTIVATION_f_df(activ, s);
		float a  = a_da.f0;
		float da = a_da.f1;
		//
		y[ty*Y__vars + depart__y + _y] =  a;
		l[ty*L__vars + depart__l + _y] = da;
	}
};

template <uint BLOQUE>
static __global__ void kerd_f_ax_b__shared_16__t(
	float * x0, uint X0_vars, uint X0, uint x0_t,
	//
	float *  y, uint Y__vars, uint  Y,
	float *  l, uint L__vars,
	//
	float * p,
	//
	uint mega_t,
	//
	uint v_x0, uint v_y, uint C0,
	//
	uint activ)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y   = thx + blockIdx.x * blockDim.x;
	uint _tc0 = thy + blockIdx.y * blockDim.y;

	uint _t = _tc0 % GRAND_T;
	uint c0 = (_tc0 - _t)/GRAND_T;

	uint depart_a0 = c0*(v_x0*v_y + v_y) + 0;
	uint depart__b = c0*(v_x0*v_y + v_y) + v_x0*v_y;
	uint depart__y = c0*v_y;
	uint depart__l = c0*v_y;
	uint depart_x0 = c0*v_x0;

	//
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );

	float s = 0;

	//	+a0@x0
	FOR(0, d, X0/BLOQUE) {
		__partage__x[thy][thx] = x0[tx0*X0_vars + depart_x0 + (d*BLOQUE + thx)];
		__partage__p[thy][thx] = p[depart_a0 + _y*X0 + (d*BLOQUE + thy)];
		__syncthreads();

	#pragma unroll
			FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
			__syncthreads();
		};

	//	+b
	#define __partage__b __partage__x[0]
	if (thy == 0) {
		__partage__b[thx] = p[depart__b + _y];
	}
	__syncthreads();
	
	s = (s + __partage__b[thx]);
	//
	f2 a_da = ACTIVATION_f_df(activ, s);
	float a  = a_da.f0;
	float da = a_da.f1;
	//
	y[ty*Y__vars + depart__y + _y] =  a;
	l[ty*L__vars + depart__l + _y] = da;
};

void dot1d_X__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	//
	uint \
		C0   =params[0], \
		activ=params[1];
	//
	uint v_x0 = inst->x_Y[0] / C0;
	uint v_y  = inst->Y      / C0;
	//
	uint x0_t = inst->x_t[0];
	uint Y    = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	uint xs_existants = x0_existe;
	//
	if (xs_existants == 1) {
		if (v_y % 16 == 0 && GRAND_T % 16 == 0) {
			kerd_f_ax_b__shared_16__t<16><<<dim3(KERD(v_y, 16), KERD(GRAND_T*C0, 16)), dim3(16, 16)>>>(
				x__d[0], inst->x_Y[0], v_x0, x0_t,
				//
				inst->y__d, inst->Y, v_y,
				inst->l__d, inst->L,
				//
				inst->p__d,
				//
				mega_t,
				//
				v_x0, v_y, C0,
				//
				activ);
		} else {
			FOR(0, c0, C0)
				kerd_f_ax_b__t<16><<<dim3(KERD(v_y, 16), KERD(GRAND_T*C0, 16)), dim3(16, 16)>>>(
					c0,
					//
					x__d[0], inst->x_Y[0], v_x0, x0_t,
					//
					inst->y__d, inst->Y, v_y,
					inst->l__d, inst->L,
					//
					inst->p__d,
					//
					mega_t,
					//
					v_x0, v_y, C0,
					//
					activ);
		};
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};