#include "hip/hip_runtime.h"
#include "dot1d_X.cuh"

uint dot1d_X__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint \
		C0   =params[0], \
		activ=params[1];
	//
	uint v_x0 = X[0] / C0;
	uint v_y  = Y    / C0;
	//
	return (v_x0*v_y + v_y) * C0;
};

uint dot1d_X__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint \
		C0   =params[0], \
		activ=params[1];
	//
	uint v_x0 = X[0] / C0;
	uint v_y  = Y    / C0;
	//
	return C0 * v_y;
};

void dot1d_X__init_poids(Inst_t * inst) {
	uint * params = inst->params;
	uint \
		C0   =params[0], \
		activ=params[1];
	//
	uint v_x0 = inst->x_Y[0] / C0;
	uint v_y  = inst->Y      / C0;
	//
	float p[inst->P];
	uint X=inst->x_Y[0], Y=inst->Y;
	FOR(0, i, inst->P) p[i] = sqrtf( 6.0 / (float)(v_x0)) * poid_1_1();

	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};