#include "hip/hip_runtime.h"
#include "pool2x2_2d.cuh"

__global__
static void kerd__pool2x2_2d(
	uint x0_t, uint X0, float * x0,
	//
	uint Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint C0, uint im_X, uint im_Y)
{
	//
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;
	uint _t = threadIdx.z + blockIdx.z * blockDim.z;
	//
	uint Y_x = im_X / 2;
	uint Y_y = im_Y / 2;
	//
	if (_y < Y_y && _x < Y_x && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		FOR(0, c0, C0) {
			float a = x0[tx0*X0 + c0*im_X*im_Y + (_y*2 + 0)*im_X + (_x*2 + 0)];
			float b = x0[tx0*X0 + c0*im_X*im_Y + (_y*2 + 0)*im_X + (_x*2 + 1)];
			float c = x0[tx0*X0 + c0*im_X*im_Y + (_y*2 + 1)*im_X + (_x*2 + 0)];
			float d = x0[tx0*X0 + c0*im_X*im_Y + (_y*2 + 1)*im_X + (_x*2 + 1)];
			y[ty*Y + c0*Y_x*Y_y + _y*Y_x + _x] = (a+b+c+d) / 4;
		}
	}
};

#define BLK 8

__global__
static void kerd__pool2x2_2d__shared(
	uint x0_t, uint X0, float * x0,
	//
	uint Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint C0, uint im_X, uint im_Y)
{
	//
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint _x = thx % im_X;	// .x
	uint c0 = (thx-_x)/im_X;// .x
	//
	uint _y = thy % im_Y;	// .y
	uint _t = (thy-_y)/im_Y;// .y
	//
	if (_x < im_X && c0 < C0 && _y < im_Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		__shared__ float im[BLK][BLK];
		//
		im[threadIdx.y][threadIdx.x] = x0[tx0*X0 + c0*im_X*im_Y + _y*im_X + _x];
		__syncthreads();

		if (threadIdx.x % 2 == 0) {
			im[threadIdx.y][threadIdx.x] = im[threadIdx.y][threadIdx.x] + im[threadIdx.y][threadIdx.x+1];
		}
		__syncthreads();

		if (threadIdx.x % 2 == 0 && threadIdx.y % 2 == 0) {
			float res = im[threadIdx.y][threadIdx.x] + im[threadIdx.y+1][threadIdx.x];
			y[ty*Y + c0*im_X/2*im_Y/2 + (_y/2)*im_X/2 + (_x/2)] = res / 4;
		}
		__syncthreads();
	}
};

void pool2x2_2d__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		C0  =params[0],	\
		im_X=params[1],	\
		im_Y=params[2];
	//
	uint Y_x = im_X / 2;
	uint Y_y = im_Y / 2;
	//
	uint X0 = inst->x_Y[0];	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	uint xs_existants = x0_existe;
	//
	if (xs_existants == 1) {
		if (true) {
			kerd__pool2x2_2d__shared<<<dim3(KERD(C0*im_X,BLK), KERD(im_Y*GRAND_T,BLK)), dim3(BLK,BLK)>>>(
				x0_t, X0, x__d[0],
				//
				inst->Y,
				inst->y__d,
				//
				ts__d, mega_t,
				//
				C0, im_X, im_Y
			);
		} else {
			kerd__pool2x2_2d<<<dim3(KERD(Y_x,8), KERD(Y_y,8), KERD(GRAND_T,8)), dim3(8,8,8)>>>(
				x0_t, X0, x__d[0],
				//
				inst->Y,
				inst->y__d,
				//
				ts__d, mega_t,
				//
				C0, im_X, im_Y
			);
		}
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};