#include "hip/hip_runtime.h"
#include "kconvl1d_stricte.cuh"

static __global__ void kerd__kconvl2d_stricte___simple(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	uint K, uint C0, uint C1, uint im_X)
{
	uint so_X = im_X;
	//
	int N = ((int)(K-1))/2;
	//
	uint _x  = threadIdx.x + blockIdx.x * blockDim.x;
	uint _c1t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint c1 = _c1t%C1;
	uint _t = (_c1t-c1)/C1;
	//
	if (_x < so_X && c1 < C1 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		//
		float ds = dy[ty*Y + c1*so_X + _x];// / (float)C0;
		//
		FOR(0, c0, C0) {
			for (int kx=-N; kx < N+1; kx++) {
				int xx = _x + kx;
				if (0 <= xx && xx < im_X) {
					//s += x0[tx0*X0 + c0*im_X + xx] * p[c1*C0*K + c0*K + (N+kx)];
					atomicAdd(&dx0[tx0*X0 + c0*im_X + xx  ], ds *  p[c1*C0*K + c0*K + (N+kx)]);
					atomicAdd(&dp [c1*C0*K + c0*K + (N+kx)], ds * x0[tx0*X0 + c0*im_X + xx  ]);
				}
			}
		}
	}
};

void kconvl1d_stricte__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3];
	//
	uint N = (K-1)/2;
	//
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__kconvl2d_stricte___simple<<<dim3(KERD((im_X),16), KERD((C1*GRAND_T),8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->p__d, inst->dp__d,
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			ts__d, mega_t,
			K, C0, C1, im_X);
	} else {
		//	rien
	}
};