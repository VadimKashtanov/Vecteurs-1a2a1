#include "kconvl1d_stricte.cuh"

uint kconvl1d_stricte__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3];
	//
	return K * C0 * C1;
};

uint kconvl1d_stricte__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

void kconvl1d_stricte__init_poids(Inst_t * inst) {
	uint * params = inst->params;
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3];
	//
	uint N = (K-1)/2;
	//
	ASSERT(N > 0);
	ASSERT(C0 > 0);
	ASSERT(C1 > 0);
	ASSERT(im_X > 0);
	//
	ASSERT(inst->x_Y[0] == C0*im_X);
	ASSERT(inst->Y      == C1*im_X);
	//
	float p[inst->P];
	uint X=inst->x_Y[0], Y=inst->Y;
	FOR(0, i, inst->P) p[i] = (1.0/(float)C0) * poid_1_1();

	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};