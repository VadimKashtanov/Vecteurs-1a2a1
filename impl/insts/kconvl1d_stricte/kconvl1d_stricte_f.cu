#include "hip/hip_runtime.h"
#include "kconvl1d_stricte.cuh"

static __global__ void kerd__kconvl2d_stricte___simple(
	uint x0_t, uint X0, float * x0,
	//
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint K, uint C0, uint C1, uint im_X)
{
	uint so_X = im_X;
	//
	int N = ((int)(K-1))/2;
	//
	uint _x  = threadIdx.x + blockIdx.x * blockDim.x;
	uint _c1t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint c1 = _c1t%C1;
	uint _t = (_c1t-c1)/C1;
	//
	if (_x < so_X && c1 < C1 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		//
		FOR(0, c0, C0) {
			for (int kx=-N; kx < N+1; kx++) {
				int xx = _x + kx;
				if (0 <= xx && xx < im_X) {
					s += x0[tx0*X0 + c0*im_X + xx] * p[c1*C0*K + c0*K + (N+kx)];
				}
			}
		}
		y[ty*Y + c1*so_X + _x] = s;// / (float)C0;
	}
};

void kconvl1d_stricte__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3];
	//
	uint N = (K-1)/2;
	//
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__kconvl2d_stricte___simple<<<dim3(KERD((im_X),16), KERD((C1*GRAND_T),8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->p__d,
			//
			inst->Y,
			inst->y__d,
			//
			ts__d, mega_t,
			K, C0, C1, im_X);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};