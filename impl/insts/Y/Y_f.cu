#include "hip/hip_runtime.h"
#include "Y.cuh"

__global__
static void kerd__Y(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		y[ty*Y + _y] = x0[tx0*X0 + _y];
	};
};

void Y__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__Y<<<dim3(KERD(Y,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};