#include "hip/hip_runtime.h"
#include "matmul1d.cuh"

static __global__ void d_kerd__matmul1d_simple_dXdP(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	uint BLOQUE_X0)
{
	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	uint __x0 = threadIdx.z + blockIdx.z * blockDim.z;

	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float _dy = dy[ty*Y + _y];
		//
		/*__shared__ float __dy__[16][16];
		if (threadIdx.z == 0) __dy__[threadIdx.y][threadIdx.x] = dy[ty*Y + _y];
		__syncthreads();
		//
		float _dy = __dy__[threadIdx.y][threadIdx.x];*/
		//
		FOR(0, i, X0 / BLOQUE_X0) {
			uint depart_x0 = __x0*(X0/BLOQUE_X0);
			//	s += x0[i]*p[y*X+i]
			atomicAdd(&dx0[tx0*X0 + depart_x0+i], _dy * p [ _y*X0 + depart_x0+i]);
			atomicAdd(&dp [ _y*X0 + depart_x0+i], _dy * x0[tx0*X0 + depart_x0+i]);
		}
	}
};

//	================================================================================
//	================================================================================
//	================================================================================

template <uint BLOQUE>
static __global__ void d_kerd__matmul1d_stricte__dX(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t)
{
	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	float s = 0;

	FOR(0, d, Y/BLOQUE) {
		__partage__x[threadIdx.y][threadIdx.x] = dy[ty*Y + (d*BLOQUE+threadIdx.x)        ];
		__partage__p[threadIdx.y][threadIdx.x] =  p[       (d*BLOQUE+threadIdx.y)*X0 + _x];
		__syncthreads();
	
	#pragma unroll
		FOR(0, i, BLOQUE) {
			s += __partage__x[threadIdx.y][i] * __partage__p[i][threadIdx.x];
		}
		__syncthreads();
	};

	atomicAdd(&dx0[tx0*X0 + _x], s);
};

template <uint BLOQUE>
static __global__ void d_kerd__matmul1d_stricte__dP(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t)
{
	//dp = x.T @ ((y-_y)*dtanh(x@p))
	
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;
	
	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	float s = 0;

	uint d = blockIdx.z;
	//FOR(0, d, T/BLOQUE) {
		uint tx0 = t_MODE((d*BLOQUE+threadIdx.y), (mega_t-x0_t));
		uint ty  = t_MODE((d*BLOQUE+threadIdx.x), (mega_t     ));
		__partage__x[threadIdx.y][threadIdx.x] = dy[ty *Y  + _y];
		__partage__p[threadIdx.y][threadIdx.x] = x0[tx0*X0 + _x];
		__syncthreads();

	#pragma unroll
		FOR(0, i, BLOQUE) {
			s += __partage__x[threadIdx.y][i] * __partage__p[i][threadIdx.x];
		}
		__syncthreads();
	//};

	atomicAdd(&dp[_y*X0 + _x], s);
};

//	-------------------------------------------------------------------------

void matmul1d__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0];	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		if (GRAND_T%16 == 0 && X0%16 == 0 && Y%16==0) {
			d_kerd__matmul1d_stricte__dX<16><<<dim3(KERD(X0, 16), KERD(GRAND_T, 16)), dim3(16, 16)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
				//
				inst->p__d, inst->dp__d,
				//
				inst->Y,
				inst->y__d, inst->dy__d,
				//
				ts__d, mega_t
			);
			d_kerd__matmul1d_stricte__dP<16><<<dim3(KERD(X0, 16), KERD(Y, 16), DIV(GRAND_T,16)), dim3(16, 16)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
				//
				inst->p__d, inst->dp__d,
				//
				inst->Y,
				inst->y__d, inst->dy__d,
				//
				ts__d, mega_t
			);
		} else {
			uint BLOQUE_X0 = 1;
			//
			//if (X0 % 16 == 0)     BLOQUE_X0 = 8;
			//else if (X0 % 8 == 0) BLOQUE_X0 = 8;
			if (X0 % 4 == 0) BLOQUE_X0 = 4;
			else if (X0 % 2 == 0) BLOQUE_X0 = 2;
			//
			d_kerd__matmul1d_simple_dXdP<<<dim3(KERD(Y, 16), KERD(GRAND_T, 16), BLOQUE_X0), dim3(16, 16)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
				//
				inst->p__d, inst->dp__d,
				//
				inst->Y,
				inst->y__d, inst->dy__d,
				//
				ts__d, mega_t,
				//
				BLOQUE_X0
			);
		}
	} else {
		//	rien
	}
};