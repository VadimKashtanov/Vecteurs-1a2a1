#include "hip/hip_runtime.h"
#include "matmul1d.cuh"

static __global__ void kerd__matmul1d___version_simple(
	uint x0_t, uint X0, float * x0,
	//
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t)
{
	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;

	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		FOR(0, i, X0) {
			//	s += x0[i]
			s += x0[tx0*X0 + i] * p [_y*X0 + i];
		}
		y[ty*Y + _y] = s;
	}
};

//	=========================== Version BLOQUE ==========================

template <uint BLOQUE>
__global__ static void kerd__matmul1d___version_stricte(
	uint x0_t, uint X0, float * x0,
	//
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t)
{
	//
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	float s = 0;
	//
	FOR(0, d, X0/BLOQUE) {
		__partage__x[threadIdx.y][threadIdx.x] = x0[tx0*X0 + (d*BLOQUE + threadIdx.x)];
		__partage__p[threadIdx.y][threadIdx.x] =  p[ _y*X0 + (d*BLOQUE + threadIdx.y)];
		__syncthreads();

	#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[threadIdx.y][i] * __partage__p[i][threadIdx.x];
		__syncthreads();
	};

	y[ty*Y + _y] = s;
};

void matmul1d__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0];	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		if (GRAND_T%16 == 0 && X0%16 == 0 && Y%16==0) {
			kerd__matmul1d___version_stricte<16><<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0],
				//
				inst->p__d,
				//
				inst->Y,
				inst->y__d,
				//
				ts__d, mega_t
			);
		} else {
			kerd__matmul1d___version_simple<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0],
				//
				inst->p__d,
				//
				inst->Y,
				inst->y__d,
				//
				ts__d, mega_t
			);
		}
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};