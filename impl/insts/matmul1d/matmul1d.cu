#include "hip/hip_runtime.h"
#include "matmul1d.cuh"

uint matmul1d__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return X[0] * Y;
};

uint matmul1d__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

void matmul1d__init_poids(Inst_t * inst) {
	float p[inst->P];
	uint X=inst->x_Y[0], Y=inst->Y;
	FOR(0, i, inst->P) p[i] = sqrtf( 6.0 / (float)(X)) * poid_1_1();

	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};