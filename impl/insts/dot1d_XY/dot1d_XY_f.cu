#include "hip/hip_runtime.h"
#include "dot1d_XY.cuh"

template <uint BLOQUE>
static __global__ void kerd_f_ax_b__t(
	uint c0,
	//
	float * x0, uint X0_vars, uint X0, uint x0_t,
	//
	float *  y, uint Y__vars, uint  Y,
	float *  l, uint L__vars,
	//
	float * p,
	//
	uint mega_t,
	//
	uint v_x0, uint v_x1, uint v_y, uint C0,
	//
	uint activ)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y   = thx + blockIdx.x * blockDim.x;
	uint /*_tc0*/_t = thy + blockIdx.y * blockDim.y;

	//uint _t = _tc0 % GRAND_T;
	//uint c0 = (_tc0 - _t)/GRAND_T;

	uint depart_a0 = c0*(v_x0*v_y + v_y) + 0;
	uint depart__b = c0*(v_x0*v_y + v_y) + v_x0*v_y;
	uint depart__y = c0*v_y;
	uint depart__l = c0*v_y;
	uint depart_x0 = c0*v_x0;

	if (_y < Y) {
		//
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );

		float s = p[depart__b + _y];
		//
		FOR(0, i, v_x0) {
			s += x0[tx0*X0_vars + depart_x0 + i] * p[depart_a0 + _y*X0 + i];
		}
		//
		float  a;
		float da;
		//
		if        (activ == 0) {
			a  = tanh(s);
			da = 1 - a*a;
		} else if (activ == 1) {
			a  = 1 / (1 + expf(-s));
			da = a * (1 - a);
		} else if (activ == 2) {
			a  = expf(-s*s);
			da = -2*s*a;
		} else if (activ == 3) {
			a  = s * (s > 0);
			da = (s > 0);
		}
		//
		y[ty*Y__vars + depart__y + _y] =  a;
		l[ty*L__vars + depart__l + _y] = da;
	}
};

template <uint BLOQUE>
static __global__ void kerd_f_ax_bx_c__t(
	uint c0,
	//
	float * x0, uint X0_vars, uint X0, uint x0_t,
	float * x1, uint X1_vars, uint X1, uint x1_t,
	//
	float *  y, uint Y__vars, uint  Y,
	float *  l, uint L__vars,
	//
	float * p,
	//
	uint mega_t,
	//
	uint v_x0, uint v_x1, uint v_y, uint C0,
	//
	uint activ)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	uint depart_a0 = c0*(v_x0*v_y + v_x1*v_y + v_y) + 0;
	uint depart_a1 = c0*(v_x0*v_y + v_x1*v_y + v_y) + v_x0*v_y;
	//
	uint depart__b = c0*(v_x0*v_y + v_x1*v_y + v_y) + v_x0*v_y + v_x1*v_y;
	//
	uint depart__y = c0*v_y;
	uint depart__l = c0*v_y;
	//
	uint depart_x0 = c0*v_x0;
	uint depart_x1 = c0*v_x1;

	if (_y < Y) {
		//
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );

		float s = p[depart__b + _y];
		//
		FOR(0, i, v_x0) {
			s += x0[tx0*X0_vars + depart_x0 + i] * p[depart_a0 + _y*X0 + i];
		}
		//
		FOR(0, i, v_x1) {
			s += x1[tx1*X1_vars + depart_x1 + i] * p[depart_a1 + _y*X1 + i];
		}
		//
		float  a;
		float da;
		//
		if        (activ == 0) {
			a  = tanh(s);
			da = 1 - a*a;
		} else if (activ == 1) {
			a  = 1 / (1 + expf(-s));
			da = a * (1 - a);
		} else if (activ == 2) {
			a  = expf(-s*s);
			da = -2*s*a;
		} else if (activ == 3) {
			a  = s * (s > 0);
			da = (s > 0);
		}
		//
		y[ty*Y__vars + depart__y + _y] =  a;
		l[ty*L__vars + depart__l + _y] = da;
	}
};

void dot1d_XY__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	//
	uint \
		C0   =params[0], \
		activ=params[1];
	//
	uint v_x0 = inst->x_Y[0] / C0;
	uint v_x1 = inst->x_Y[1] / C0;
	uint v_y  = inst->Y      / C0;
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	uint Y    = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	uint xs_existants = x0_existe + x1_existe;
	//
	if (xs_existants == 2) {
		FOR(0, c0, C0)
			kerd_f_ax_bx_c__t<16><<<dim3(KERD(v_y, 16), KERD(GRAND_T, 16)), dim3(16, 16)>>>(
				c0,
				//
				x__d[0], inst->x_Y[0], v_x0, x0_t,
				x__d[1], inst->x_Y[1], v_x1, x1_t,
				//
				inst->y__d, inst->Y, v_y,
				inst->l__d, inst->L,
				//
				inst->p__d,
				//
				mega_t,
				//
				v_x0, v_x1, v_y, C0,
				//
				activ);
	} else if (xs_existants == 1) {
		uint i0 = (x0_existe ? 0 : 1);
		//
		FOR(0, c0, C0) {
			kerd_f_ax_b__t<16><<<dim3(KERD(v_y, 16), KERD(GRAND_T, 16)), dim3(16, 16)>>>(
				c0,
				//
				x__d[i0], inst->x_Y[i0], inst->x_Y[i0]/C0, inst->x_t[i0],
				//
				inst->y__d, inst->Y, v_y,
				inst->l__d, inst->L,
				//
				inst->p__d,
				//
				mega_t,
				//
				v_x0, v_x1, v_y, C0,
				//
				activ);
		}
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};