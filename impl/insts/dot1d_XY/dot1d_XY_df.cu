#include "hip/hip_runtime.h"
#include "dot1d_XY.cuh"

template <uint BLOQUE>
static __global__ void d_kerd_f_ax_b__t(
	uint c0,
	//
	float * x0, uint X0_vars, uint X0, uint x0_t, float * dx0,
	//
	float *  y, uint Y__vars, uint  Y, float * dy,
	float *  l, uint L__vars,
	//
	float * p, float * dp,
	//
	uint mega_t,
	//
	uint v_x0, uint v_x1, uint v_y, uint C0,
	//
	uint activ)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y   = thx + blockIdx.x * blockDim.x;
	uint /*_tc0*/_t = thy + blockIdx.y * blockDim.y;

	//uint _t = _tc0 % GRAND_T;
	//uint c0 = (_tc0 - _t)/GRAND_T;

	uint depart_a0 = c0*(v_x0*v_y + v_y) + 0;
	uint depart__b = c0*(v_x0*v_y + v_y) + v_x0*v_y;
	uint depart__y = c0*v_y;
	uint depart__l = c0*v_y;
	uint depart_x0 = c0*v_x0;

	if (_y < Y && _t < GRAND_T) {
		//
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float _dy = dy[ty*Y__vars + depart__y + _y];
		float  _l =  l[ty*L__vars + depart__l + _y];
		//
		float ds = _dy * _l;
		//
		atomicAdd(&dp[depart__b + _y], ds);
		//
		FOR(0, i, v_x0) {
			//s += x0[tx0*X0_vars + depart_x0 + i] * p[depart_a0 + _y*X0 + i];
			atomicAdd(&dx0[tx0*X0_vars + depart_x0 + i], ds*p[depart_a0 + _y*X0 + i]);
			atomicAdd(&dp [depart_a0  + _y*X0      + i], ds*x0[tx0*X0_vars + depart_x0 + i]);
		}
	}
};

template <uint BLOQUE>
static __global__ void d_kerd_f_ax_bx_c__t(
	uint c0,
	//
	float * x0, uint X0_vars, uint X0, uint x0_t, float * dx0,
	float * x1, uint X1_vars, uint X1, uint x1_t, float * dx1,
	//
	float *  y, uint Y__vars, uint  Y, float * dy,
	float *  l, uint L__vars,
	//
	float * p, float * dp,
	//
	uint mega_t,
	//
	uint v_x0, uint v_x1, uint v_y, uint C0,
	//
	uint activ)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	uint depart_a0 = c0*(v_x0*v_y + v_x1*v_y + v_y) + 0;
	uint depart_a1 = c0*(v_x0*v_y + v_x1*v_y + v_y) + v_x0*v_y;
	//
	uint depart__b = c0*(v_x0*v_y + v_x1*v_y + v_y) + v_x0*v_y + v_x1*v_y;
	//
	uint depart__y = c0*v_y;
	uint depart__l = c0*v_y;
	//
	uint depart_x0 = c0*v_x0;
	uint depart_x1 = c0*v_x1;

	if (_y < Y && _t < GRAND_T) {
		//
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float _dy = dy[ty*Y__vars + depart__y + _y];
		float  _l =  l[ty*L__vars + depart__l + _y];
		//
		float ds = _dy * _l;
		//
		atomicAdd(&dp[depart__b + _y], ds);
		//
		FOR(0, i, v_x0) {
			//s += x0[tx0*X0_vars + depart_x0 + i] * p[depart_a0 + _y*X0 + i];
			atomicAdd(&dx0[tx0*X0_vars + depart_x0 + i], ds*p[depart_a0 + _y*X0 + i]);
			atomicAdd(&dp [depart_a0  + _y*X0      + i], ds*x0[tx0*X0_vars + depart_x0 + i]);
		}

		FOR(0, i, v_x1) {
			//s += x1[tx1*X1_vars + depart_x1 + i] * p[depart_a1 + _y*X1 + i];
			atomicAdd(&dx1[tx1*X1_vars + depart_x1 + i], ds*p[depart_a1 + _y*X1 + i]);
			atomicAdd(&dp [depart_a1  + _y*X1      + i], ds*x1[tx1*X1_vars + depart_x1 + i]);
		}
	}
};

void dot1d_XY__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	//
	uint \
		C0   =params[0], \
		activ=params[1];
	//
	uint v_x0 = inst->x_Y[0] / C0;
	uint v_x1 = inst->x_Y[1] / C0;
	uint v_y  = inst->Y      / C0;
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	uint Y    = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	uint xs_existants = x0_existe + x1_existe;
	//
	if (xs_existants == 2) {
		FOR(0, c0, C0)
			d_kerd_f_ax_bx_c__t<16><<<dim3(KERD(v_y, 16), KERD(GRAND_T, 16)), dim3(16, 16)>>>(
				c0,
				//
				x__d[0], inst->x_Y[0], v_x0, x0_t, dx__d[0],
				x__d[1], inst->x_Y[1], v_x1, x1_t, dx__d[1],
				//
				inst->y__d, inst->Y, v_y, inst->dy__d,
				inst->l__d, inst->L,
				//
				inst->p__d, inst->dp__d,
				//
				mega_t,
				//
				v_x0, v_x1, v_y, C0,
				//
				activ);
	} else if (xs_existants == 1) {
		uint i0 = (x0_existe ? 0 : 1);
		//
		FOR(0, c0, C0) {
			d_kerd_f_ax_b__t<16><<<dim3(KERD(v_y, 16), KERD(GRAND_T, 16)), dim3(16, 16)>>>(
				c0,
				//
				x__d[i0], inst->x_Y[i0], inst->x_Y[i0]/C0, inst->x_t[i0], dx__d[i0],
				//
				inst->y__d, inst->Y, v_y, inst->dy__d,
				inst->l__d, inst->L,
				//
				inst->p__d, inst->dp__d,
				//
				mega_t,
				//
				v_x0, v_x1, v_y, C0,
				//
				activ);
		}
	} else {
		//	rien
	}
};