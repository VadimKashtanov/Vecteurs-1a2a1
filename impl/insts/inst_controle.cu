#include "hip/hip_runtime.h"
#include "insts.cuh"

__global__
static void kerd_inst_zero_mega_t(float * y, uint Y, uint mega_t)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		y[t_MODE(_t, mega_t)*Y + _y] = 0.0;
	};
};

void inst_zero_mega_t(Inst_t * inst, uint mega_t) {
	//kerd_inst_zero_mega_t<<<DIM2(inst->Y, GRAND_T, 16,16)>>>(
	kerd_inst_zero_mega_t<<<dim3(KERD(inst->Y,16), KERD(GRAND_T,16)),dim3(16,16)>>>(
		inst->y__d,
		inst->Y,
		mega_t
	);
};