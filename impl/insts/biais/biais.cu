#include "hip/hip_runtime.h"
#include "biais.cuh"

uint biais__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return Y;
};

uint biais__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

void biais__init_poids(Inst_t * inst) {
	float p[inst->P];
	uint Y=inst->Y;
	FOR(0, i, inst->P) p[i] = sqrtf( 6.0 / (float)(Y)) * poid_1_1();//(2*rnd()-1);

	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};