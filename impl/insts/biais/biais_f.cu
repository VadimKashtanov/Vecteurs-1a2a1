#include "hip/hip_runtime.h"
#include "biais.cuh"

__global__
static void kerd__biais(
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint mega_t)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;
	//
#define _y thx
#define _t thy
	//
	if (_y < Y && _t < GRAND_T) {
		uint ty  = t_MODE(_t, mega_t);
		
		y[ty*Y + _y] = p[_y];
	};
};

void biais__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint Y = inst->Y;
	kerd__biais<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
		inst->p__d,
		//
		inst->Y,
		inst->y__d,
		//
		mega_t
	);
};