#include "hip/hip_runtime.h"
#include "cadrans_pondérés.cuh"

static __global__ void kerd__cadrans_ponderes__simple(
	uint x0_t, uint X0, float * x0,
	//
	float * p,
	//
	uint    Y, uint    L,
	float * y, float * l,
	//
	uint * ts__d, uint mega_t,
	//
	uint Cx, uint C0, uint C1)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	thx = Cx*C1
	uint _cx = thx % Cx;
	uint _c1 = (thx-_cx)/Cx;

	//	thy = GRAND_T
	uint _t = thy;

	if (_cx < Cx && _c1 < C1 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		uint pos_y = ty*Y + _c1*Cx + _cx;
		//
		//float normalisation = 0;
		//
		FOR(0, _c0, C0) {
			uint pos_x0 = tx0*C0*Cx + _c0*Cx + _cx;
			uint pos_p  = _c1*C0*Cx + _cx*C0 + _c0;
			//
			//normalisation += p[pos_p];
			//
			s += x0[pos_x0] * p[pos_p];
		}
		//normalisation=1;//
		//l[2*pos_y+0] = normalisation;
		//l[2*pos_y+1] =             s;
		y[pos_y] = s;// / normalisation;
	}
};

//	---------------------------------------------------------------------------------

void cadrans_pondérés__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		Cx=params[0],	\
		C0=params[1],	\
		C1=params[2];
	//
	uint x0_t = inst->x_t[0];
	uint Y    = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__cadrans_ponderes__simple<<<dim3(KERD((Cx*C1),16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->p__d,
			//
			inst->Y,    inst->L,
			inst->y__d, inst->l__d,
			//
			ts__d, mega_t,
			//
			Cx, C0, C1
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};