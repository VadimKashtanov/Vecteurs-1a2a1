#include "cadrans_pondérés.cuh"

uint cadrans_pondérés__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint \
		Cx=params[0],	\
		C0=params[1],	\
		C1=params[2];
	//
	return C1 * C0 * Cx;
};

uint cadrans_pondérés__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint \
		Cx=params[0],	\
		C0=params[1],	\
		C1=params[2];
	return 0;//2 * Cx * C1;
};

void cadrans_pondérés__init_poids(Inst_t * inst) {
	uint * params = inst->params;
	uint \
		Cx=params[0],	\
		C0=params[1],	\
		C1=params[2];
	//
	ASSERT(inst->Y == C1 * Cx);
	ASSERT(inst->x_Y[0] == C0 * Cx);
	//
	float p[inst->P];
	FOR(0, i, inst->P) p[i] = (1.0/(float)C0) * poid_1_1();
	//
	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};