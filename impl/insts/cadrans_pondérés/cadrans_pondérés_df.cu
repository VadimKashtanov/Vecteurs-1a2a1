#include "hip/hip_runtime.h"
#include "cadrans_pondérés.cuh"

static __global__ void d_kerd__cadrans_ponderes__simple(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,             uint    L,
	float * y, float * dy, float * l,
	//
	uint * ts__d, uint mega_t,
	//
	uint Cx, uint C0, uint C1)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	thx = Cx*C1
	uint _cx = thx % Cx;
	uint _c1 = (thx-_cx)/Cx;

	//	thy = GRAND_T
	uint _t = thy;

	if (_cx < Cx && _c1 < C1 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint pos_y = ty*Y + _c1*Cx + _cx;
		//
		//float normalisation =  l[2*pos_y+0];
		//float s             =  l[2*pos_y+1];
		//
		float _dy           = dy[pos_y];
		//
		float ds = _dy;// / normalisation;
		//float dn = -_dy * s / (normalisation*normalisation);
		//
		FOR(0, _c0, C0) {
			uint pos_x0 = tx0*C0*Cx + _c0*Cx + _cx;
			uint pos_p  = _c1*C0*Cx + _cx*C0 + _c0;
			//
			//normalisation += p[pos_p];
			//atomicAdd(&dp[pos_p], dn);
			//
			//s += x0[pos_x0] * p[pos_p];
			atomicAdd(&dx0[pos_x0],  p[pos_p ] * ds);
			atomicAdd(&dp [pos_p ], x0[pos_x0] * ds);
		};
	}
};

//	---------------------------------------------------------------------------------

void cadrans_pondérés__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		Cx=params[0],	\
		C0=params[1],	\
		C1=params[2];
	//
	uint x0_t = inst->x_t[0];
	uint Y    = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		d_kerd__cadrans_ponderes__simple<<<dim3(KERD((Cx*C1),16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->p__d, inst->dp__d,
			//
			inst->Y,                 inst->L,
			inst->y__d, inst->dy__d, inst->l__d,
			//
			ts__d, mega_t,
			//
			Cx, C0, C1
		);
	} else {
		//inst_zero_mega_t(inst, mega_t);
	}
};