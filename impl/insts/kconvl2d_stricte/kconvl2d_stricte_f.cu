#include "hip/hip_runtime.h"
#include "kconvl2d_stricte.cuh"

static __global__ void kerd__kconvl2d_stricte___simple(
	uint x0_t, uint X0, float * x0,
	//
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint K, uint C0, uint C1, uint im_X, uint im_Y)
{
	uint so_X = im_X;
	uint so_Y = im_Y;
	//
	int N = ((int)(K-1))/2;
	//
	uint _xy  = threadIdx.x + blockIdx.x * blockDim.x;
	uint _c1t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint _x = _xy%im_X;
	uint _y = (_xy-_x)/im_X;
	//
	uint c1 = _c1t%C1;
	uint _t = (_c1t-c1)/C1;
	//
	if (_y < so_Y && _x < so_X && c1 < C1 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		//
		FOR(0, c0, C0) {
			for (int kx=-N; kx < N+1; kx++) {
				for (int ky=-N; ky < N+1; ky++) {
					int xx = _x + kx;
					int xy = _y + ky;
					if (0 <= xx && xx < im_X && 0 <= xy && xy < im_Y) {
						s += x0[tx0*X0 + c0*im_X*im_Y + xy*im_X + xx] * p[c1*C0*K*K + c0*K*K + (N+ky)*K + (N+kx)];
					}
				}
			}
		}
		y[ty*Y + c1*so_X*so_Y + _y*so_X + _x] = s;// / C0;
	}
};

static __global__ void kerd__kconvl2d_stricte___c0_thz_8(
	uint x0_t, uint X0, float * x0,
	//
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint K, uint C0, uint C1, uint im_X, uint im_Y)
{
	uint so_X = im_X;
	uint so_Y = im_Y;
	//
	int N = ((int)(K-1))/2;
	//
	uint _xy  = threadIdx.x + blockIdx.x * blockDim.x;
	uint _c1t = threadIdx.y + blockIdx.y * blockDim.y;
	uint thz  = threadIdx.z;
	//
	uint _x = _xy%im_X;
	uint _y = (_xy-_x)/im_X;
	//
	uint c1 = _c1t%C1;
	uint _t = (_c1t-c1)/C1;
	//
	if (_y < so_Y && _x < so_X && c1 < C1 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		//
		__shared__ float __y[16][8];
		if (thz == 0) __y[threadIdx.y][threadIdx.x] = 0;
		__syncthreads();
		//
		FOR(0, _c0, C0/8) {
			uint c0 = _c0*8 + thz;
			for (int kx=-N; kx < N+1; kx++) {
				for (int ky=-N; ky < N+1; ky++) {
					int xx = _x + kx;
					int xy = _y + ky;
					if (0 <= xx && xx < im_X && 0 <= xy && xy < im_Y) {
						s += x0[tx0*X0 + c0*im_X*im_Y + xy*im_X + xx] * p[c1*K*K*C0 + c0*K*K + (N+ky)*K + (N+kx)];
					}
				}
			}
			atomicAdd(&__y[threadIdx.y][threadIdx.x], s);
			__syncthreads();
		}

		if (thz == 0) y[ty*Y + c1*so_X*so_Y + _y*so_X + _x] = __y[threadIdx.y][threadIdx.x];
	}
};

void kconvl2d_stricte__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3],
		im_Y=params[4];
	//
	uint N = (K-1)/2;
	//
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		if (C0 % 8 == 0 && false) {
			kerd__kconvl2d_stricte___c0_thz_8<<<dim3(KERD((im_X*im_Y),16), KERD((C1*GRAND_T),8), 1), dim3(16,8,8)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0],
				//
				inst->p__d,
				//é
				inst->Y,
				inst->y__d,
				//
				ts__d, mega_t,
				K, C0, C1, im_X, im_Y);
		} else {
			kerd__kconvl2d_stricte___simple<<<dim3(KERD((im_X*im_Y),16), KERD((C1*GRAND_T),8)), dim3(16,8)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0],
				//
				inst->p__d,
				//
				inst->Y,
				inst->y__d,
				//
				ts__d, mega_t,
				K, C0, C1, im_X, im_Y);
		}
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};