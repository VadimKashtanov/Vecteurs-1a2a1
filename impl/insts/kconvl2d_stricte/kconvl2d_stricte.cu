#include "kconvl2d_stricte.cuh"

uint kconvl2d_stricte__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3],
		im_Y=params[4];
	//
	return K * K * C0 * C1;
};

uint kconvl2d_stricte__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

void kconvl2d_stricte__init_poids(Inst_t * inst) {
	uint * params = inst->params;
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3],
		im_Y=params[4];
	//
	uint N = (K-1)/2;
	//
	ASSERT(N > 0);
	ASSERT(C0 > 0);
	ASSERT(C1 > 0);
	ASSERT(im_X > 0);
	ASSERT(im_Y > 0);
	//
	ASSERT(inst->x_Y[0] == C0*im_X*im_Y);
	ASSERT(inst->Y    == C1*im_X*im_Y);
	//
	float p[inst->P];
	uint X=inst->x_Y[0], Y=inst->Y;
	FOR(0, i, inst->P) p[i] = 1.0/(float)C0 * poid_1_1();

	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};