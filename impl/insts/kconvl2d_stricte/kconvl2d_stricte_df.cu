#include "hip/hip_runtime.h"
#include "kconvl2d_stricte.cuh"

static __global__ void d_kerd__kconvl2d_stricte___simple(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	uint K, uint C0, uint C1, uint im_X, uint im_Y)
{
	uint so_X = im_X;
	uint so_Y = im_Y;
	//
	int N = ((int)(K-1))/2;
	//
	uint _xy  = threadIdx.x + blockIdx.x * blockDim.x;
	uint _c1t = threadIdx.y + blockIdx.y * blockDim.y;
	uint   c0 = threadIdx.z + blockIdx.z * blockDim.z;
	//
	uint _x = _xy%im_X;
	uint _y = (_xy-_x)/im_X;
	//
	uint c1 = _c1t%C1;
	uint _t = (_c1t-c1)/C1;
	//
	if (_y < so_Y && _x < so_X && c1 < C1 && _t < GRAND_T && c0 < C0) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		__shared__ float __dy[8][8];
		if (threadIdx.z == 0) __dy[threadIdx.y][threadIdx.x] = dy[ty*Y + c1*so_X*so_Y + _y*so_X + _x];
		__syncthreads();
		//float _dy = dy[ty*Y + c1*so_X*so_Y + _y*so_X + _x];
		float _dy = __dy[threadIdx.y][threadIdx.x];// / (float)C0;
		//
		//FOR(0, c0, C0) {
		for (int kx=-N; kx < N+1; kx++) {
			for (int ky=-N; ky < N+1; ky++) {
				int xx = _x + kx;
				int xy = _y + ky;
				if (0 <= xx && xx < im_X && 0 <= xy && xy < im_Y) {
					//s += x0[tx0*X0 + c0*im_X*im_Y + xy*im_X + xx] * p[c1*K*K*C0 + c0*K*K + (N+ky)*K + (N+kx)];
					uint pos_x0 = tx0*X0 + c0*im_X*im_Y + xy*im_X + xx;
					uint pos_p  = c1*K*K*C0 + c0*K*K + (N+ky)*K + (N+kx);
					atomicAdd(&dx0[pos_x0], _dy *      1     * p[pos_p]);
					atomicAdd(&dp [pos_p ], _dy * x0[pos_x0] *     1   );
				}
			}
		}
	}
};

void kconvl2d_stricte__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3],
		im_Y=params[4];
	//
	uint N = (K-1)/2;
	//
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	//printf("### K=%i C0=%i C1=%i im_X=%i im_Y=%i\n", K, C0, C1, im_X, im_X);
	//
	if (x0_existe) {
		uint _KC0 = 8;
		if (C0 < _KC0) _KC0 = 1;
		//
		d_kerd__kconvl2d_stricte___simple<<<dim3(KERD((im_X*im_Y),8), KERD((C1*GRAND_T),8), KERD(C0,_KC0)), dim3(8,8,_KC0)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->p__d, inst->dp__d,
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			ts__d, mega_t,
			K, C0, C1, im_X, im_Y
		);
	} else {
		//	inst_zero_mega_t(inst, mega_t);
		//	rien
	}
};