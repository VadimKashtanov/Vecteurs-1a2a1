#include "hip/hip_runtime.h"
#include "matmul1d_canal.cuh"

uint matmul1d_canal__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint \
		C0=params[0],	\
		C1=params[1],	\
		M =params[2];
	//
	uint v_x = X[0] / C0;
	uint v_y = Y / C1;
	//
	return v_x * v_y * (C0/M) * (C1/M)*M;
};

uint matmul1d_canal__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	//uint \
		C0 =params[0],	\
		C1=params[1],	\
		M =params[2];
	return 0;
};

void matmul1d_canal__init_poids(Inst_t * inst) {
	uint * params = inst->params;
	uint \
		C0 =params[0],	\
		C1=params[1],	\
		M =params[2];
	//
	uint v_x = inst->x_Y[0] / C0;
	uint v_y = inst->Y / C1;
	//
	float p[inst->P];
	FOR(0, i, inst->P) p[i] = sqrtf( 6.0 / (float)(v_x*C0/M)) * poid_1_1();
	//
	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};