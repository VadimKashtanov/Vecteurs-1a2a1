#include "hip/hip_runtime.h"
#include "matmul1d_canal.cuh"

static __global__ void d_kerd__matmul1d_canal__simple(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	uint _c0, uint _c1, uint v_x, uint v_y, uint M, uint C0, uint C1)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	v_y*_c1
	uint __v_y = thx % v_y;
	uint ___c1 = (thx-__v_y)/v_y;

	//	GRAND_T*M
	uint _t = thy % GRAND_T;
	uint _m = (thy-_t)/GRAND_T;

	if (__v_y < v_y && ___c1 < _c1 && _t < GRAND_T && _m < M) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint pos_y = _m*_c1*v_y + ___c1*v_y + __v_y;
		//
		float _dy = dy[ty*Y + pos_y];
		//
		FOR(0, __c0, _c0) {
			//	mat mul v_x;v_y
			FOR(0, i, v_x) {
				//	s += x0[i]
				//s += x0[tx0*X0 + _m*_c0*v_x + __c0*v_x + i] * p[(_m*_c1*v_y + ___c1*v_y + __v_y)*v_x + __c0*v_x + i];
				uint pos_x0 = tx0*X0 + _m*_c0*v_x + __c0*v_x + i;
				uint pos_p  = pos_y*v_x*_c0 + __c0*v_x + i;
				//
				atomicAdd(&dx0[pos_x0], _dy *  p[pos_p ]);
				atomicAdd(&dp [pos_p ], _dy * x0[pos_x0]);
			}

		}
	}
};

//	====================================================================================

/*template <uint BLOQUE>
static __global__ void d_kerd__matmul1d_canal_stricte__dX(
	uint _m, uint ___c1, uint ___c0,
	//
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	uint _c0, uint _c1, uint v_x, uint v_y, uint M, uint C0, uint C1)
{
	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	float s = 0;

	FOR(0, d, v_y/BLOQUE) {
		__partage__x[threadIdx.y][threadIdx.x] = dy[ty*Y + _m*_c1*v_y + ___c1*v_y + (d*BLOQUE+threadIdx.x)];
		__partage__p[threadIdx.y][threadIdx.x] =  p[(_m*_c1*v_y + ___c1*v_y + (d*BLOQUE+threadIdx.y))* + v_x*_c0 + ___c0*v_x + _x];
		__syncthreads();
	
	#pragma unroll
		FOR(0, i, BLOQUE) {
			s += __partage__x[threadIdx.y][i] * __partage__p[i][threadIdx.x];
		}
		__syncthreads();
	};

	atomicAdd(&dx0[tx0*X0 + _m*_c0*v_x + ___c0*v_x + _x], s);
};

template <uint BLOQUE>
static __global__ void d_kerd__matmul1d_canal_stricte__dP(
	uint _m, uint ___c1, uint ___c0,
	//
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	uint _c0, uint _c1, uint v_x, uint v_y, uint M, uint C0, uint C1)
{
	//dp = x.T @ ((y-_y)*dtanh(x@p))
	
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;
	
	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	float s = 0;

	uint d = blockIdx.z;
	//FOR(0, d, T/BLOQUE) {
		uint tx0 = t_MODE((d*BLOQUE+threadIdx.y), (mega_t-x0_t));
		uint ty  = t_MODE((d*BLOQUE+threadIdx.x), (mega_t     ));
		//
		__partage__x[threadIdx.y][threadIdx.x] = dy[ty * Y  + _m*_c1*v_y + ___c1*v_y + _y];
		__partage__p[threadIdx.y][threadIdx.x] = x0[tx0*X0 + _m*_c0*v_x + ___c0*v_x + _x];
		__syncthreads();

	#pragma unroll
		FOR(0, i, BLOQUE) {
			s += __partage__x[threadIdx.y][i] * __partage__p[i][threadIdx.x];
		}
		__syncthreads();
	//};

	uint pos_y = _m*_c1*v_y + ___c1*v_y + _y;
	uint pos_p = pos_y*v_x*_c0 + ___c0*v_x + _x;
	atomicAdd(&dp[pos_p], s);
};*/

//	------------------------------------------------------------------------------------

void matmul1d_canal__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		C0=params[0],	\
		C1=params[1],	\
		M =params[2];
	//
	uint _c0 = C0 / M;
	uint _c1 = C1 / M;
	//
	uint v_x = inst->x_Y[0] / C0;
	uint v_y = inst->Y      / C1;
	//
	uint x0_t = inst->x_t[0];
	uint Y    = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		/*if (v_x%16==0 && v_y%16==0 && GRAND_T%16==0) {
			FOR(0, ___c0, _c0) {
				FOR(0, ___c1, _c1) {
					FOR(0, _m, M) {
						d_kerd__matmul1d_canal_stricte__dX<16><<<dim3(KERD(v_x, 16), KERD(GRAND_T, 16)), dim3(16, 16)>>>(
							_m, ___c1, ___c0,
							//
							inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
							//
							inst->p__d, inst->dp__d,
							//
							inst->Y,
							inst->y__d, inst->dy__d,
							//
							ts__d, mega_t,
							//
							_c0, _c1, v_x, v_y, M, C0, C1
						);
						d_kerd__matmul1d_canal_stricte__dP<16><<<dim3(KERD(v_x, 16), KERD(v_y, 16), DIV(GRAND_T,16)), dim3(16, 16)>>>(
							_m, ___c1, ___c0,
							//
							inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
							//
							inst->p__d, inst->dp__d,
							//
							inst->Y,
							inst->y__d, inst->dy__d,
							//
							ts__d, mega_t,
							//
							_c0, _c1, v_x, v_y, M, C0, C1
						);
					}
				}
			}
		} else*/if (true) {
			d_kerd__matmul1d_canal__simple<<<dim3(KERD((v_y*_c1),16), KERD((GRAND_T*M),16)), dim3(16,16)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
				//
				inst->p__d, inst->dp__d,
				//
				inst->Y,
				inst->y__d, inst->dy__d,
				//
				ts__d, mega_t,
				//
				_c0, _c1, v_x, v_y, M, C0, C1
			);
		}
	} else {
		//	inst_zero_mega_t(inst, mega_t);
	}
};