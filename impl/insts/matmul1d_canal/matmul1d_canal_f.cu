#include "hip/hip_runtime.h"
#include "matmul1d_canal.cuh"

static __global__ void kerd__matmul1d_canal__simple(
	uint x0_t, uint X0, float * x0,
	//
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint _c0, uint _c1, uint v_x, uint v_y, uint M, uint C0, uint C1)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	v_y*_c1
	uint __v_y = thx % v_y;
	uint ___c1 = (thx-__v_y)/v_y;

	//	GRAND_T*M
	uint _t = thy % GRAND_T;
	uint _m = (thy-_t)/GRAND_T;

	if (__v_y < v_y && ___c1 < _c1 && _t < GRAND_T && _m < M) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		uint pos_y = _m*_c1*v_y + ___c1*v_y + __v_y;
		FOR(0, __c0, _c0) {
			//	mat mul v_x;v_y
			FOR(0, i, v_x) {
				//	s += x0[i]
				uint pos_x0 = tx0*X0 + _m*_c0*v_x + __c0*v_x + i;
				uint pos_p  = pos_y*v_x*_c0 + __c0*v_x + i;
				//
				s += x0[pos_x0] * p[pos_p];
			}

		}
		//printf("%i %f\n", ty*Y + _m*_c1*v_y + ___c1*v_y + __v_y, s);
		y[ty*Y + pos_y] = s;
	}
};

//	---------------------------------------------------------------------------------

void matmul1d_canal__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		C0=params[0],	\
		C1=params[1],	\
		M =params[2];
	//
	uint _c0 = C0 / M;
	uint _c1 = C1 / M;
	//
	uint v_x = inst->x_Y[0] / C0;
	uint v_y = inst->Y      / C1;
	//
	uint x0_t = inst->x_t[0];
	uint Y    = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		if (true) {
			kerd__matmul1d_canal__simple<<<dim3(KERD((v_y*_c1),16), KERD((GRAND_T*M),16)), dim3(16,16)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0],
				//
				inst->p__d,
				//
				inst->Y,
				inst->y__d,
				//
				ts__d, mega_t,
				//
				_c0, _c1, v_x, v_y, M, C0, C1
			);
		}
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};