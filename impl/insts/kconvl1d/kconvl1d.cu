#include "kconvl1d.cuh"

uint kconvl1d__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3],
		im_Y=params[4];
	//
	return K * C0 * C1;
};

uint kconvl1d__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

void kconvl1d__init_poids(Inst_t * inst) {
	uint * params = inst->params;
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3],
		im_Y=params[4];
	//
	uint N = (K-1)/2;
	//
	uint X=inst->x_Y[0], Y=inst->Y;
	//
	ASSERT(   N > 0);
	ASSERT(  C0 > 0);
	ASSERT(  C1 > 0);
	ASSERT(im_X > 0);
	//
	ASSERT(im_Y == im_X-N-N);
	//
	ASSERT(inst->x_Y[0] == C0*im_X);
	ASSERT(inst->Y      == C1*im_Y);
	//
	float p[inst->P];
	FOR(0, i, inst->P) p[i] = (1.0/(float)C0) * poid_1_1();
	//
	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};