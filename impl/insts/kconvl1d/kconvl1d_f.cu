#include "hip/hip_runtime.h"
#include "kconvl1d.cuh"

static __global__ void kerd__kconvl2d___simple(
	uint x0_t, uint X0, float * x0,
	//
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint K, uint C0, uint C1, uint im_X, uint im_Y)
{
	int N = ((int)(K-1))/2;
	//
	uint _y  = threadIdx.x + blockIdx.x * blockDim.x;
	uint _c1t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint c1 = _c1t%C1;
	uint _t = (_c1t-c1)/C1;
	//
	if (_y < im_Y && c1 < C1 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		//
		FOR(0, c0, C0) {
			for (int kx=-N; kx < N+1; kx++) {
				int xx = N + _y + kx;
				assert(0 <= xx && xx < im_X);
				s += x0[tx0*X0 + c0*im_X + xx] * p[c1*C0*K + c0*K + (N+kx)];
			}
		}
		y[ty*Y + c1*im_Y + _y] = s;// / (float)C0;
	}
};

void kconvl1d__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		K   =params[0],
		C0  =params[1],
		C1  =params[2],
		im_X=params[3],
		im_Y=params[4];
	//
	uint N = (K-1)/2;
	//
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__kconvl2d___simple<<<dim3(KERD((im_Y),16), KERD((C1*GRAND_T),8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->p__d,
			//
			inst->Y,
			inst->y__d,
			//
			ts__d, mega_t,
			K, C0, C1, im_X, im_Y);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};