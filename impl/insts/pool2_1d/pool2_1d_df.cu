#include "hip/hip_runtime.h"
#include "pool2_1d.cuh"

__global__
static void d_kerd__pool2_1d(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	uint C0, uint im_X)
{
	//
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint Y_x = im_X / 2;
	//
	if (_x < Y_x && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		FOR(0, c0, C0) {
			//float a = x0[tx0*X0 + c0*im_X + (_x*2 + 0)];
			//float b = x0[tx0*X0 + c0*im_X + (_x*2 + 1)];
			float ds = dy[ty*Y + c0*Y_x + _x] / 2;
			atomicAdd(&dx0[tx0*X0 + c0*im_X + (_x*2 + 0)], ds);
			atomicAdd(&dx0[tx0*X0 + c0*im_X + (_x*2 + 1)], ds);
		}
	}
};

void pool2_1d__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		C0  =params[0],	\
		im_X=params[1];
	//
	uint Y_x = im_X / 2;
	//
	uint X0 = inst->x_Y[0];	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	uint xs_existants = x0_existe;
	//
	if (x0_existe) {
		d_kerd__pool2_1d<<<dim3(KERD(Y_x,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			x0_t, X0, x__d[0], dx__d[0],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			ts__d, mega_t,
			//
			C0, im_X
		);
	} else {
		//	rien
	}
};