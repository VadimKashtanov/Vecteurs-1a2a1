#include "hip/hip_runtime.h"
#include "const.cuh"

__global__
static void kerd__const(
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	float constante)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;
	//
#define _y thx
#define _t thy
	//
	if (_y < Y && _t < GRAND_T) {
		uint ty  = t_MODE(_t, mega_t);
		
		y[ty*Y + _y] = constante;
	};
};

void const__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint Y = inst->Y;
	//
	float constante = (float)inst->params[0];
	//
	kerd__const<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
		inst->Y,
		inst->y__d,
		//
		mega_t,
		//
		constante
	);
};