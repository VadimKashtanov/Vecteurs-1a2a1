#include "hip/hip_runtime.h"
#include "somme3.cuh"

__global__
static void d_kerd__somme1(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t)
{
	//
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		//y[ty*Y + _y] = x0[tx0*X0 + _y];
		float __dy = dy[ty*Y + _y];
		atomicAdd(&dx0[tx0*X0 + _y], 1 * __dy);
	}
};

__global__
static void d_kerd__somme2(
	uint x0_t, uint X0, float * x0, float * dx0,
	uint x1_t, uint X1, float * x1, float * dx1,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t)
{
	//
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		//y[ty*Y + _y] = x0[tx0*X0 + _y] + x1[tx1*X1 + _y];
		float __dy = dy[ty*Y + _y];
		atomicAdd(&dx0[tx0*X0 + _y], 1 * __dy);
		atomicAdd(&dx1[tx1*X1 + _y], 1 * __dy);
	}
};

__global__
static void d_kerd__somme3(
	uint x0_t, uint X0, float * x0, float * dx0,
	uint x1_t, uint X1, float * x1, float * dx1,
	uint x2_t, uint X2, float * x2, float * dx2,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t)
{
	//
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint tx2 = t_MODE(_t, mega_t-x2_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		//y[ty*Y + _y] = x0[tx0*X0 + _y] + x1[tx1*X1 + _y] + x2[tx2*X2 + _y];
		float __dy = dy[ty*Y + _y];
		atomicAdd(&dx0[tx0*X0 + _y], 1 * __dy);
		atomicAdd(&dx1[tx1*X1 + _y], 1 * __dy);
		atomicAdd(&dx2[tx2*X2 + _y], 1 * __dy);
	}
};

void somme3__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0];	uint x0_t = inst->x_t[0];
	uint X1 = inst->x_Y[1];	uint x1_t = inst->x_t[1];
	uint X2 = inst->x_Y[2];	uint x2_t = inst->x_t[2];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	bool x2_existe = (mega_t != 0 ? true : (x2_t != 1));
	//
	uint xs_existants = x0_existe + x1_existe + x2_existe;
	//
	if (xs_existants == 3) {
		uint existe[3] = {x0_existe, x1_existe, x2_existe};
		//
		uint _i0 = INDEX3(existe[0], existe[1], existe[2], 1);
		FOR(0, i, 3) if (i != _i0 && existe[i] != 0) existe[i] += 1;
		uint _i1 = INDEX3(existe[0], existe[1], existe[2], 2);
		FOR(0, i, 3) if (i != _i1 && existe[i] != 0) existe[i] += 1;
		uint _i2 = INDEX3(existe[0], existe[1], existe[2], 3);
		//
		d_kerd__somme3<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[_i0], inst->x_Y[_i0], x__d[_i0], dx__d[_i0],
			inst->x_t[_i1], inst->x_Y[_i1], x__d[_i1], dx__d[_i1],
			inst->x_t[_i2], inst->x_Y[_i2], x__d[_i2], dx__d[_i2],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			ts__d, mega_t
		);
	} else if (xs_existants == 2) {
		uint existe[3] = {x0_existe, x1_existe, x2_existe};
		//
		uint _i0 = INDEX3(existe[0], existe[1], existe[2], 1);
		FOR(0, i, 3) if (i != _i0 && existe[i] != 0) existe[i] += 1;
		uint _i1 = INDEX3(existe[0], existe[1], existe[2], 2);
		//
		d_kerd__somme2<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[_i0], inst->x_Y[_i0], x__d[_i0], dx__d[_i0],
			inst->x_t[_i1], inst->x_Y[_i1], x__d[_i1], dx__d[_i1],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			ts__d, mega_t
		);
	} else if (xs_existants == 1) {
		uint existe[3] = {x0_existe, x1_existe, x2_existe};
		//
		uint _i0 = INDEX3(existe[0], existe[1], existe[2], 1);
		//
		d_kerd__somme1<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[_i0], inst->x_Y[_i0], x__d[_i0], dx__d[_i0],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			ts__d, mega_t
		);
	} else if (xs_existants == 0) {
		//	inst_zero_mega_t(inst, mega_t);
	}
};